
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <malloc.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

#define BLOCK_SIZE 16
#define BLOCK_SCALE 256

struct axis {
	float x;
	float y;
	float z;
	float v;
};

int cal(FILE *fp);

int main(int argc, char *argv[])
{
	FILE *fp;

	if (argc >= 2) {
		fp = fopen(argv[1], "r");

	} else {
		printf("Usage: prog datafile\n");
		exit(1);
	}

	cal(fp);
	fclose(fp);

	return 0;
}

__device__ float distance(struct axis *p1, struct axis *p2)
{
	return (p1->x - p2->x) * (p1->x - p2->x) +
		(p1->y - p2->y) * (p1->y - p2->y) +
		(p1->z - p2->z) * (p1->z - p2->z);
}

__global__ void threadCode(int count, float radius2, struct axis *points, float *result)
{
	int thId;
	int i;

	thId = blockIdx.x + blockIdx.y*blockDim.x;
	thId = threadIdx.x + threadIdx.y*BLOCK_SIZE + thId*BLOCK_SCALE;

	if (thId >= count)
		return;

	for (i = thId + 1; i < count; i++) {
		if (distance(&points[thId], &points[i]) <= radius2) {
			result[thId] += points[i].v;
			result[i] += points[thId].v;
		}
	}
}

int paralize(int count, float radius, struct axis *points, float *sum)
{
	struct axis *cudaPtr;
	float *cudaRst;

	hipMalloc((void **)&cudaPtr, sizeof(struct axis)*count);
	hipMalloc((void **)&cudaRst, sizeof(float)*count);

	hipMemcpy(cudaPtr, points, sizeof(struct axis)*count, hipMemcpyHostToDevice);
	hipMemset(cudaRst, 0x0, sizeof(float)*count);

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid((count+BLOCK_SCALE-1)/BLOCK_SCALE, 1);

	threadCode<<<dimGrid, dimBlock>>>(count, radius*radius, cudaPtr, cudaRst);
	
	hipMemcpy(sum, cudaRst, sizeof(float)*count, hipMemcpyDeviceToHost);
	hipFree(cudaPtr);
	hipFree(cudaRst);
	return 0;
}

int cal(FILE *fp)
{
	int point_count;
	float radius;
	struct axis *points;
	int i;
	struct timeval tv_start, tv_end;
	double time_cost;
	float *sum;

	fscanf(fp, "%d %f", &point_count, &radius);

	points = (struct axis *) malloc(sizeof(struct axis)*point_count);
	sum = (float *) malloc(sizeof(float)*point_count);
	memset((void *) sum, 0x0, sizeof(float)*point_count);

	for (i = 0; i < point_count; i++)
		fscanf(fp, "%f %f %f %f", &points[i].x,	&points[i].y, 
				&points[i].z, &points[i].v);

	/* execute calculation and get time stamp */
	gettimeofday(&tv_start, NULL);
	paralize(point_count, radius, points, sum);
	gettimeofday(&tv_end, NULL);

	time_cost = 1000000 * (tv_end.tv_sec - tv_start.tv_sec) +
		(tv_end.tv_usec - tv_start.tv_usec);
	time_cost /= 1000000;

	/* output the result and time cost */
	for (i = 0; i < point_count; i++)
		printf("Point %5d: %f\n", i+1, sum[i]);
	printf("Time: %lf\n", time_cost);

	free(points);
	free(sum);
	return 0;
}
